#define FP double

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>

// A: n * p, B: p * m, C: n * m
__global__ void gpu_matrixmult(FP *a, FP *b, FP *c, int n, int p, int m) {
  int col = threadIdx.x + blockDim.x * blockIdx.x;
  int row = threadIdx.y + blockDim.y * blockIdx.y;

  int indexb = col;
  int index = row * m + col;
  
  if (col < m && row < n) {
    c[index] = 0.;
    for (int indexa = row * p; indexa < (row * p + p); indexa++, indexb += m) 
      c[index] += a[indexa] * b[indexb];
  }
}

int main(int argc, char *argv[]) {
  int i, j; // loop counters

  int gpucount = 0; // Count of available GPUs
  int gpunum = 0; // Device number to use
  int Grid_Dim_x = 1; // Grid dimension, x 
  int Grid_Dim_y = 1; // Grid dimension, y
  int Block_Dim_x = 1; // Block dimension, x 
  int Block_Dim_y = 1; // Block dimension, y

  int n, p, m; // matrix dimension (A: n * p, B: p * m, C: n * m)
  FP *a, *b, *c;
  FP *dev_a, *dev_b, *dev_c;
  int sizeA, sizeB, sizeC; // number of bytes in arrays

  hipEvent_t start, stop; // using cuda events to measure time
  float elapsed_time_ms; // which is applicable for asynchronous code also
  hipError_t errorcode;

  // -------------------- SET PARAMETERS AND DATA -----------------------

  errorcode = hipGetDeviceCount(&gpucount);
  if (errorcode == hipErrorNoDevice) {
    printf("No GPUs are visible\n");
    exit(-1);
  }
  else {
    printf("Device count = %d\n", gpucount);
  }

  if (argc != 5) {
    printf("Usage: matmul-1a <matrix dim n> <matrix dim p> <matrix dim m> <block dim> \n");
    exit (-1);  
  }

  n = atoi(argv[1]);
  p = atoi(argv[2]);
  m = atoi(argv[3]);

  Block_Dim_x = atoi(argv[4]); // Square block
  Block_Dim_y = Block_Dim_x;
  if (Block_Dim_x * Block_Dim_y > 1024) {
    printf("Error, too many threads in block\n");
    exit (-1);
  }

  Grid_Dim_x = m / Block_Dim_x;
  Grid_Dim_y = n / Block_Dim_y;
  if (Grid_Dim_x * Block_Dim_x < m || Grid_Dim_y * Block_Dim_y < n) {
    printf("Error, number of threads in x/y dimensions less than number of array elements\n");
    exit(-1);
  }

  hipSetDevice(gpunum);
  printf("Using device %d\n", gpunum);
  
  printf("Matrix Dimension = A (%d, %d), B (%d, %d), C (%d, %d) \n", n, p, p, m, n, m);
  printf("Block_Dim = (%d, %d), Grid_Dim = (%d, %d) \n", Block_Dim_x, Block_Dim_y, Grid_Dim_x, Grid_Dim_y);

  dim3 Grid(Grid_Dim_x, Grid_Dim_y); // Grid structure
  dim3 Block(Block_Dim_x, Block_Dim_y); // Block structure

  sizeA = n * p * sizeof(FP);
  sizeB = p * m * sizeof(FP);
  sizeC = n * m * sizeof(FP);

  a = (FP *) malloc(sizeA); // dynamically allocated memory for arrays on host
  b = (FP *) malloc(sizeB);
  c = (FP *) malloc(sizeC); // results from GPU

  srand(12345);
  for (i = 0; i < n; i++) {
    for (j = 0; j < p; j++) {
      a[i * p + j] = (FP) rand() / (FP) RAND_MAX;
    }
  }

  for (i = 0; i < p; i++) {
    for (j = 0; j < m; j++) {
      b[i * m + j] = (FP) rand() / (FP) RAND_MAX;
    }
  }

  // ------------- COMPUTATION DONE ON GPU ----------------------------

  hipMalloc((void**) &dev_a, sizeA); // allocate memory on device
  hipMalloc((void**) &dev_b, sizeB);
  hipMalloc((void**) &dev_c, sizeC);

  hipMemcpy(dev_a, a, sizeA, hipMemcpyHostToDevice); // copy from CPU tp GPU
  hipMemcpy(dev_b, b, sizeB, hipMemcpyHostToDevice);

  hipEventCreate(&start); // instrument code to measure start time
  hipEventCreate(&stop);
  
  hipEventRecord(start, 0);
  // cudaEventSynchronize(start); // not needed

  gpu_matrixmult<<<Grid, Block>>>(dev_a, dev_b, dev_c, n, p, m);

  hipEventRecord(stop, 0); // instrument code to measure end time
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop);

  hipMemcpy(c, dev_c, sizeC, hipMemcpyDeviceToHost); // copy from GPU to CPU

  printf("Time to calculate results on GPU: %f ms.\n", elapsed_time_ms); // exec. time

  // ----------------------------- clean up ------------------------------

  free(a);
  free(b);
  free(c);
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}


