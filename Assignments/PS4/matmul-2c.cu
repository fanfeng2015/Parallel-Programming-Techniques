#define FP float

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>

// A: n * p, B: p * m, C: n * m
__global__ void gpu_matrixmult(FP *a, FP *b, FP *c, int n, int p, int m, int TW) {
  extern __shared__ FP bigarray[];
  FP *atile = &bigarray[0], *btile = &bigarray[TW * TW];

  int tx = threadIdx.x; int ty = threadIdx.y; FP cvalue = 0.;
  // current thread is responsible for computing the entry at (row, col) of C
  int col = tx + blockDim.x * blockIdx.x;
  int row = ty + blockDim.y * blockIdx.y;

  for (int tile = 0; tile < p / TW; tile++) {
    int indexa = row * p + tile * TW + tx;   // current thread caches this entry of A
    int indexb = (tile * TW + ty) * m + col; // current thread caches this entry of B

    atile[ty * TW + tx] = (tile * TW + tx < p && row < n) ? a[indexa] : 0.;
    btile[ty * TW + tx] = (tile * TW + ty < p && col < m) ? b[indexb] : 0.;
    __syncthreads(); // make sure tiles are loaded before all threads can use them

    for (int k = 0; k < TW; k++) cvalue += atile[ty * TW + k] * btile[k * TW + tx];
    __syncthreads(); // make sure we’re done using the tiles before overwriting them
  }

  if (col < m && row < n) {
    c[row * m + col] = cvalue;
  }
}

int main(int argc, char *argv[]) {
  int i, j; // loop counters

  int gpucount = 0; // Count of available GPUs
  int gpunum = 0; // Device number to use
  int Grid_Dim_x = 1; // Grid dimension, x 
  int Grid_Dim_y = 1; // Grid dimension, y
  int Block_Dim_x = 1; // Block dimension, x 
  int Block_Dim_y = 1; // Block dimension, y
  int TW;

  int n, p, m; // matrix dimension (A: n * p, B: p * m, C: n * m)
  FP *a, *b, *c;
  FP *dev_a, *dev_b, *dev_c;
  size_t sizeA, sizeB, sizeC; // number of bytes in arrays

  hipEvent_t start, stop; // using cuda events to measure time
  float elapsed_time_ms; // which is applicable for asynchronous code also
  hipError_t errorcode;

  // -------------------- SET PARAMETERS AND DATA -----------------------

  errorcode = hipGetDeviceCount(&gpucount);
  if (errorcode == hipErrorNoDevice) {
    printf("No GPUs are visible\n");
    exit(-1);
  }
  else {
    printf("Device count = %d\n", gpucount);
  }

  if (argc != 5) {
    printf("Usage: matmul-1a <matrix dim n> <matrix dim p> <matrix dim m> <block dim> \n");
    exit (-1);  
  }

  n = atoi(argv[1]);
  p = atoi(argv[2]);
  m = atoi(argv[3]);

  Block_Dim_x = atoi(argv[4]); // Square block
  Block_Dim_y = Block_Dim_x;
  TW = Block_Dim_x; // Simplest case: TW = Block_Dim_x = Block_Dim_y
  if (Block_Dim_x * Block_Dim_y > 1024) {
    printf("Error, too many threads in block\n");
    exit (-1);
  }

  Grid_Dim_x = (m - 1) / Block_Dim_x + 1;
  Grid_Dim_y = (n - 1) / Block_Dim_y + 1;
  if (Grid_Dim_x * Block_Dim_x < m || Grid_Dim_y * Block_Dim_y < n) {
    printf("Error, number of threads in x/y dimensions less than number of array elements\n");
    exit(-1);
  }

  hipSetDevice(gpunum);
  printf("Using device %d\n", gpunum);
  
  printf("Matrix Dimension = A (%d, %d), B (%d, %d), C (%d, %d) \n", n, p, p, m, n, m);
  printf("Block_Dim = (%d, %d), Grid_Dim = (%d, %d) \n", Block_Dim_x, Block_Dim_y, Grid_Dim_x, Grid_Dim_y);

  dim3 Grid(Grid_Dim_x, Grid_Dim_y); // Grid structure
  dim3 Block(Block_Dim_x, Block_Dim_y); // Block structure

  sizeA = n * p * sizeof(FP);
  sizeB = p * m * sizeof(FP);
  sizeC = n * m * sizeof(FP);

  a = (FP *) malloc(sizeA); // dynamically allocated memory for arrays on host
  b = (FP *) malloc(sizeB);
  c = (FP *) malloc(sizeC); // results from GPU

  srand(12345);
  for (i = 0; i < n; i++) {
    for (j = 0; j < p; j++) {
      a[i * p + j] = (FP) rand() / (FP) RAND_MAX;
    }
  }

  for (i = 0; i < p; i++) {
    for (j = 0; j < m; j++) {
      b[i * m + j] = (FP) rand() / (FP) RAND_MAX;
    }
  }

  // ------------- COMPUTATION DONE ON GPU ----------------------------

  hipMalloc((void**) &dev_a, sizeA); // allocate memory on device
  hipMalloc((void**) &dev_b, sizeB);
  hipMalloc((void**) &dev_c, sizeC);

  hipMemcpy(dev_a, a, sizeA, hipMemcpyHostToDevice); // copy from CPU tp GPU
  hipMemcpy(dev_b, b, sizeB, hipMemcpyHostToDevice);

  hipEventCreate(&start); // instrument code to measure start time
  hipEventCreate(&stop);
  
  hipEventRecord(start, 0);
  // cudaEventSynchronize(start); // not needed

  size_t Ns = 2 * TW * TW * sizeof(FP); // amount of shared momery
  gpu_matrixmult<<<Grid, Block, Ns>>>(dev_a, dev_b, dev_c, n, p, m, TW);

  hipEventRecord(stop, 0); // instrument code to measure end time
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time_ms, start, stop);

  hipMemcpy(c, dev_c, sizeC, hipMemcpyDeviceToHost); // copy from GPU to CPU

  printf("Time to calculate results on GPU: %f ms.\n", elapsed_time_ms); // exec. time

  // ----------------------------- clean up ------------------------------

  free(a);
  free(b);
  free(c);
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}


